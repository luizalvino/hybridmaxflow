#include "hip/hip_runtime.h"


#include <stdio.h>

#define BLOCKSIZE 512

//#define DEBUG

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/***********************************************/
/* MEMCPY1 - EACH THREAD COPIES ONE FLOAT ONLY */
/***********************************************/
__global__ void memcpy1(float *src, float *dst, unsigned int N)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < N) {
        float a0 = src[tid];
        dst[tid] = a0;
    }
}

/*******************************************/
/* MEMCPY2 - EACH THREAD COPIES TWO FLOATS */
/*******************************************/
__global__ void memcpy2(float *src, float *dst, unsigned int N)
{
    const int tid = threadIdx.x + blockIdx.x * (2 * blockDim.x);

    if (tid < N) {
        float a0 = src[tid];
        float a1 = src[tid + blockDim.x];
        dst[tid] = a0;
        dst[tid + blockDim.x] = a1;
    }

}

/********************************************/
/* MEMCPY4 - EACH THREAD COPIES FOUR FLOATS */
/********************************************/
__global__ void memcpy4(float *src, float *dst, unsigned int N)
{
    const int tid = threadIdx.x + blockIdx.x * (4 * blockDim.x);

    if (tid < N) {

        float a0 = src[tid];
        float a1 = src[tid + blockDim.x];
        float a2 = src[tid + 2 * blockDim.x];
        float a3 = src[tid + 3 * blockDim.x];

        dst[tid] = a0;
        dst[tid + blockDim.x] = a1;
        dst[tid + 2 * blockDim.x] = a2;
        dst[tid + 3 * blockDim.x] = a3;

    }

}

/***********************************************/
/* MEMCPY4_2 - EACH THREAD COPIES FOUR FLOATS2 */
/***********************************************/
__global__ void memcpy4_2(float2 *src, float2 *dst, unsigned int N)
{
    const int tid = threadIdx.x + blockIdx.x * (4 * blockDim.x);

    if (tid < N/2) {

        float2 a0 = src[tid];
        float2 a1 = src[tid + blockDim.x];
        float2 a2 = src[tid + 2 * blockDim.x];
        float2 a3 = src[tid + 3 * blockDim.x];

        dst[tid] = a0;
        dst[tid + blockDim.x] = a1;
        dst[tid + 2 * blockDim.x] = a2;
        dst[tid + 3 * blockDim.x] = a3;

    }

}

/********/
/* MAIN */
/********/
int main()
{
    const int N = 131072;

    const int N_iter = 20;

    // --- Setting host data and memory space for result
    float* h_vect   = (float*)malloc(N*sizeof(float));
    float* h_result = (float*)malloc(N*sizeof(float));
    for (int i=0; i<N; i++) h_vect[i] = i;

    // --- Setting device data and memory space for result
    float* d_src;  gpuErrchk(hipMalloc((void**)&d_src,  N*sizeof(float)));
    float* d_dest1; gpuErrchk(hipMalloc((void**)&d_dest1, N*sizeof(float)));
    float* d_dest2; gpuErrchk(hipMalloc((void**)&d_dest2, N*sizeof(float)));
    float* d_dest4; gpuErrchk(hipMalloc((void**)&d_dest4, N*sizeof(float)));
    float* d_dest4_2; gpuErrchk(hipMalloc((void**)&d_dest4_2, N*sizeof(float)));
    gpuErrchk(hipMemcpy(d_src, h_vect, N*sizeof(float), hipMemcpyHostToDevice));

    // --- Warmup
    for (int i=0; i<N_iter; i++) memcpy1<<<iDivUp(N,BLOCKSIZE), BLOCKSIZE>>>(d_src, d_dest1, N);

    // --- Creating events for timing
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /***********/
    /* MEMCPY1 */
    /***********/
    hipEventRecord(start, 0);
    for (int i=0; i<N_iter; i++) {
        memcpy1<<<iDivUp(N,BLOCKSIZE), BLOCKSIZE>>>(d_src, d_dest1, N);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GB/s = %f\n", (1.e-6)*(float)(N*N_iter*sizeof(float))/time);
    gpuErrchk(hipMemcpy(h_result, d_dest1, N*sizeof(int), hipMemcpyDeviceToHost));

    /***********/
    /* MEMCPY2 */
    /***********/
    hipEventRecord(start, 0);
    for (int i=0; i<N_iter; i++) {
        memcpy2<<<iDivUp(N/2,BLOCKSIZE), BLOCKSIZE>>>(d_src, d_dest2, N);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GB/s = %f\n", (1.e-6)*(float)(N*N_iter*sizeof(float))/time);
    gpuErrchk(hipMemcpy(h_result, d_dest2, N*sizeof(int), hipMemcpyDeviceToHost));

    /***********/
    /* MEMCPY4 */
    /***********/
    hipEventRecord(start, 0);
    for (int i=0; i<N_iter; i++) {
        memcpy4<<<iDivUp(N/4,BLOCKSIZE), BLOCKSIZE>>>(d_src, d_dest4, N);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GB/s = %f\n", (1.e-6)*(float)(N*N_iter*sizeof(float))/time);
    gpuErrchk(hipMemcpy(h_result, d_dest4, N*sizeof(int), hipMemcpyDeviceToHost));


    /*************/
    /* MEMCPY4_2 */
    /*************/
    hipEventRecord(start, 0);
    for (int i=0; i<N_iter; i++) {
        memcpy4_2<<<iDivUp(N/8,BLOCKSIZE), BLOCKSIZE>>>((float2*)d_src, (float2*)d_dest4_2, N);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GB/s = %f\n", (1.e-6)*(float)(N*N_iter*sizeof(float))/time);
    gpuErrchk(hipMemcpy(h_result, d_dest4_2, N*sizeof(int), hipMemcpyDeviceToHost));


    hipDeviceReset();
    return 0;
}